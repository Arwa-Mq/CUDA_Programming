#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdio>
#include <vector>
#include <string>
#include <iostream>
#include <hip/hip_runtime.h>

#include "cosmosis/datablock/datablock.hh"

using namespace std;
using namespace cosmosis;

__global__
void boost_factor_kernel(const double* R, double* B, double rs, double b0, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    double x = R[i] / rs;
    double fx = 0.0;

    if (x > 1.0) {
        fx = atan(sqrt(x * x - 1.0)) / sqrt(x * x - 1.0);
    } else if (fabs(x - 1.0) < 1e-8) {
        fx = 1.0;
    } else {
        fx = atanh(sqrt(1.0 - x * x)) / sqrt(1.0 - x * x);
    }

    double denom = x * x - 1.0;
    if (fabs(denom) < 1e-10) denom = 1e-10;

    double B_i = 1.0 + b0 * (1.0 - fx) / denom;
    if (isnan(B_i)) {
        B_i = (b0 + 3.0) / 3.0;
    }

    B[i] = B_i;
}

void run_boost_factor_gpu(const vector<double>& R_cpu, double rs, double b0, vector<double>& B_out) {
    int N = R_cpu.size();
    double *R_d, *B_d;

    hipMalloc(&R_d, N * sizeof(double));
    hipMalloc(&B_d, N * sizeof(double));

    hipMemcpy(R_d, R_cpu.data(), N * sizeof(double), hipMemcpyHostToDevice);

    int threads = 128;
    int blocks = (N + threads - 1) / threads;
    boost_factor_kernel<<<blocks, threads>>>(R_d, B_d, rs, b0, N);

    hipMemcpy(B_out.data(), B_d, N * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(R_d);
    hipFree(B_d);
}

int setup(DataBlock* options, DataBlock* config) {
    // Read data from disk
    string path = "/global/cfs/cdirs/des/jesteves/data/boost_factor/y1/profiles";
    FILE* fp = fopen((path + "/full-unblind-v2-mcal-zmix_y1clust_l3_z0_zpdf_boost.dat").c_str(), "r");
    FILE* fpcov = fopen((path + "/full-unblind-v2-mcal-zmix_y1clust_l3_z0_zpdf_boost_cov.dat").c_str(), "r");

    if (!fp || !fpcov) {
        cerr << "ERROR: Data file not found.\n";
        return 1;
    }

    vector<double> R, data_vector;
    double r_tmp, b_tmp, sigma_tmp;
    for (int i = 0; i < 8; i++) {
        fscanf(fp, "%lf %lf %lf", &r_tmp, &b_tmp, &sigma_tmp);
        R.push_back(r_tmp);
        data_vector.push_back(b_tmp);
    }

    double cov[8][8];
    for (int i = 0; i < 8; ++i)
        for (int j = 0; j < 8; ++j)
            fscanf(fpcov, "%lf", &cov[i][j]);

    fclose(fp);
    fclose(fpcov);

    config->put_vector("boost", "R", R);
    config->put_vector("boost", "data", data_vector);
    for (int i = 0; i < 8; i++)
        for (int j = 0; j < 8; j++)
            config->put_double("boost_cov", "C_" + to_string(i) + "_" + to_string(j), cov[i][j]);

    return 0;
}

int execute(DataBlock* block, DataBlock* config) {
    vector<double> R, data;
    config->get_array("boost", "R", R);
    config->get_array("boost", "data", data);

    double logrs, logb0;
    block->get_val("Boost_Factor_Model_Values", "logrs_30", logrs);
    block->get_val("Boost_Factor_Model_Values", "logb0_30", logb0);

    double rs = pow(10.0, logrs);
    double b0 = pow(10.0, logb0);

    vector<double> model(R.size());
    run_boost_factor_gpu(R, rs, b0, model);

    // Covariance
    double cov[8][8];
    for (int i = 0; i < 8; i++)
        for (int j = 0; j < 8; j++)
            config->get_val("boost_cov", "C_" + to_string(i) + "_" + to_string(j), cov[i][j]);

    // Chi-squared
    double chisq = 0.0;
    for (int i = 0; i < 8; ++i)
        for (int j = 0; j < 8; ++j)
            chisq += (model[i] - data[i]) * cov[i][j] * (model[j] - data[j]);

    double logL = -0.5 * chisq;
    block->put_double("likelihoods", "boost_factor_likelihood_like", logL);

    return 0;
}

int cleanup(DataBlock* config) {
    return 0;
}
